#include "hip/hip_runtime.h"
#ifdef _WIN32
#include <windows.h>
#endif

#include <thrust/copy.h>
#include <thrust/device_vector.h>

#include <cuda_gl_interop.h>

#include "vtkScalarsToColors.h"
#include "vtkPistonDataObject.h"
#include "vtkPistonDataWrangling.h"
#include "vtkPistonScalarsColors.h"
#include "vtkPistonMinMax.h"
#include "vtkPistonReference.h"

#include "vtkgl.h"

#include <iostream>

using namespace std;

namespace vtkpiston {

bool AlmostEqualRelativeAndAbs(float A, float B,
            float maxDiff, float maxRelDiff)
  {
    // Check if the numbers are really close -- needed
    // when comparing numbers near zero.
    float diff = fabs(A - B);
    if (diff <= maxDiff)
        return true;

    A = fabs(A);
    B = fabs(B);
    float largest = (B > A) ? B : A;

    if (diff <= largest * maxRelDiff)
        return true;
    return false;
  }

template <typename ValueType>
struct color_map : thrust::unary_function<ValueType, float3>
{
    const ValueType min;
    const ValueType max;
    const int size;
    float *table;
    const int numberOfChanels;

    color_map(float *rtable, int arrSize, int noOfChanels,
      ValueType rMin, ValueType rMax) :
      min(rMin),
      max(rMax),
      size((arrSize / noOfChanels) - 1),
      table(rtable),
      numberOfChanels(noOfChanels)
      {
      }

    __host__ __device__
    float3 operator()(ValueType val)
    {
      int index = 0;
      if((max - min) > 0.0)
        {
        index = ( (val - min) / (max - min) ) * size;
        }

      if (index < 0) index = 0;
      if (index > size) index = size;
      index *= numberOfChanels;

      float3 color;
      if(numberOfChanels == 1)
        {
        color = make_float3(table[index], table[index], table[index]);
        }
      else if(numberOfChanels == 2)
        {
        color = make_float3(table[index], table[index + 1], 0.0f);
        }
      else if(numberOfChanels == 3)
        {
        color = make_float3(table[index], table[index + 1], table[index + 2]);
        }
      else
        {
        // Not supported
        }

      return color;
    }
};

//------------------------------------------------------------------------------
void CudaGLInit()
{
  hipDeviceProp_t prop;
  int dev;

  // Fill it with zeros
  memset(&prop,0,sizeof(hipDeviceProp_t));

  // Pick a GPU capable of 1.0 or better
  prop.major=1; prop.minor=0;
  hipChooseDevice(&dev,&prop);

  // Set OpenGL device
  hipError_t res = cudaGLSetGLDevice(dev);

  if (res != hipSuccess)
    {
    cerr << "Set device failed  ... " << hipGetErrorString(res) << endl;
    return;
    }
}

//------------------------------------------------------------------------------
void CudaRegisterBuffer(struct hipGraphicsResource **vboResource,
                        GLuint vboBuffer)
{
  hipError_t res =
    hipGraphicsGLRegisterBuffer(vboResource, vboBuffer,
                                cudaGraphicsMapFlagsWriteDiscard);
  if (res != hipSuccess)
  {
    cerr << "Register buffer failed ... " << hipGetErrorString(res) << endl;
    return;
  }
}

//------------------------------------------------------------------------------
void CudaUnregisterResource(struct hipGraphicsResource *vboResource)
{
  hipError_t res = hipGraphicsUnregisterResource(vboResource);
  if (res != hipSuccess)
  {
    cerr << "Unregister buffer failed ... " << hipGetErrorString(res) << endl;
    return;
  }
}

//------------------------------------------------------------------------------
void CudaTransferToGL(vtkPistonDataObject *id, vtkMTimeType dataObjectMTimeCache,
                      vtkPistonScalarsColors *psc,
                      hipGraphicsResource **vboResources,
                      bool &hasNormals, bool &hasColors)
{
  vtkPistonReference *tr = id->GetReference();
  if (tr->type != VTK_POLY_DATA || tr->data == NULL)
    {
    // Type mismatch, don't bother trying
    return;
    }

  vtk_polydata *pD = (vtk_polydata *)tr->data;

  // Claim access to buffer for cuda
  hipError_t res;
  res = hipGraphicsMapResources(3, vboResources, 0);
  if (res != hipSuccess)
  {
    cerr << "Claim for CUDA failed ... " << hipGetErrorString(res) << endl;
    return;
  }

  size_t num_bytes;
  float *vertexBufferData, *normalsBufferData;
  float3 *colorsBufferData;
  res = hipGraphicsResourceGetMappedPointer
      ((void **)&vertexBufferData, &num_bytes, vboResources[0]);
  if(res != hipSuccess)
  {
    cerr << "Get mappedpointer for vertices failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }
  res = hipGraphicsResourceGetMappedPointer
      ((void **)&normalsBufferData, &num_bytes, vboResources[1]);
  if(res != hipSuccess)
  {
    cerr << "Get mappedpointer for normals failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }
  res = hipGraphicsResourceGetMappedPointer
      ((void **)&colorsBufferData, &num_bytes, vboResources[2]);
  if(res != hipSuccess)
  {
    cerr << "Get mappedpointer for colors failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }

  // Copy on card verts to the shared on card gl buffer
  thrust::copy(pD->points->begin(), pD->points->end(),
               thrust::device_ptr<float>(vertexBufferData));

  hasNormals = false;
  if (pD->normals)
    {
    hasNormals = true;

    // Copy on card verts to the shared on card gl buffer
    thrust::copy(pD->normals->begin(), pD->normals->end(),
                 thrust::device_ptr<float>(normalsBufferData));
    }
  hasColors = false;

  if (pD->scalars)
    {
    double scalarRange[2];
    id->GetScalarsRange(scalarRange);

    hasColors = true;

    if(id->GetMTime() > dataObjectMTimeCache)
      {
      vtkPiston::minmax_pair<float> result = vtkPiston::find_min_max(
                                              pD->scalars);

      scalarRange[0] = static_cast<double>(result.min_val);
      scalarRange[1] = static_cast<double>(result.max_val);

      // Set parameters to compute scalars colors
      const int numvalues = 256;
      id->SetScalarsRange(scalarRange);
      psc->SetTableRange(scalarRange[0], scalarRange[1]);
      psc->SetNumberOfValues(numvalues);
      }

    std::vector<float> *colors = psc->ComputeScalarsColorsf(VTK_RGB);

    // Copy to GPU
    thrust::device_vector<float> onGPU(colors->begin(), colors->end());
    float *raw_ptr = thrust::raw_pointer_cast(&onGPU[0]);

    // Now run each scalar data through the map to choose a color for it

    // \NOTE: Since GPU most likely going to calculate range using single
    // floating point precision, we may lose precision and hence, we need
    // to check if the range min and max are almost equal
    //TODO: Remove this when piston gives us exactly same values for
    //isocontour.
    float tempRange[2] =
      {
      static_cast<float>(scalarRange[0]),
      static_cast<float>(scalarRange[1])
      };
    if( AlmostEqualRelativeAndAbs(scalarRange[0], scalarRange[1],
                                  numeric_limits<float>::epsilon(),
                                  numeric_limits<float>::epsilon() * 10) )
      {
      tempRange[1] = tempRange[0]+1.0;
      }

    color_map<float> colorMap(raw_ptr, onGPU.size(), VTK_RGB, tempRange[0], tempRange[1]);
    thrust::copy(thrust::make_transform_iterator(pD->scalars->begin(),
                                                 colorMap),
                 thrust::make_transform_iterator(pD->scalars->end(), colorMap),
                 thrust::device_ptr<float3>(colorsBufferData));
    }

  // Allow GL to access again
  res = hipGraphicsUnmapResources(3, vboResources, 0);
  if (res != hipSuccess)
  {
    cerr << "Release from CUDA failed ... " << hipGetErrorString(res) << endl;
    return;
  }

  return;
}

} //namespace
